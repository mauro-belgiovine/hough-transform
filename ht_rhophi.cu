#include "hip/hip_runtime.h"
//
//  ht_helix.cpp
//  
//
//  Created by Lorenzo Rinaldi on 29/04/14.
//
//
// compile:
// nvcc -I/usr/local/cuda-5.5/samples/common/inc -I/usr/local/cuda-5.5/targets/x86_64-linux/include -gencode arch=compute_20,code=sm_21 -o ht_rhophi ht_rhophi.cu

#include <hip/hip_runtime.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions 

#include "simpleIndexing.cu"

#include <string.h>
#include <cmath>
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <unistd.h>

using namespace std;

#define NHMAX 300
#define Nsec 4 // Numero settori in piano trasverso
#define Ntheta 16 // Numero settori in piano longitudinale
#define Nphi 1024 // Numero bin angolo polare
#define Nrho 1024 // Numero bin distanza radiale

#define rhomin 500.f // mm
#define rhomax 100000.f // mm
#define phimin 0.f // rad
#define phimax 2*M_PI // rad
#define thetamin 0.f // rad
#define thetamax M_PI // rad

#define ac_soglia 4 // soglia nella matrice di accumulazione

/* --- DEFINE TO ALTER EXECUTION --- */
//#define PARALLEL_REDUX_MAX
//#define VERBOSE_DUMP
#define CUDA_MALLOCHOST_OUTPUT
#define CUDA_MANAGED_TRANSFER

#define max_tracks_out 100

int acc_Mat [ Nsec ][ Ntheta ][Nphi ] [Nrho ];
//int Max_rel [ Nsec ][ Ntheta ][Nphi ] [Nrho ];
int debug_accMat[ Nsec ][ Ntheta ][ Nphi ] [ Nrho ];

float dtheta= M_PI/Ntheta;
float drho= (rhomax-rhomin)/Nrho;
float dphi= (phimax-phimin)/Nphi;

vector<float> x_values;
vector<float> y_values;
vector<float> z_values;

#ifndef PARALLEL_REDUX_MAX

struct track_param{
      int acc;
      /*unsigned int isec;
      unsigned int ith;
      unsigned int iphi;
      unsigned int irho;*/
    };
    
#ifndef CUDA_MALLOCHOST_OUTPUT
struct track_param host_out_tracks[ Nsec * Ntheta * Nphi * Nrho ];
#endif

#endif

//lock definition
#ifndef __LOCK_H__
#define __LOCK_H__

struct Lock {
    int *mutex;
    Lock( void ) {
         hipMalloc( (void**)&mutex, sizeof(int) ) ;
         hipMemset( mutex, 0, sizeof(int) );
    }

    ~Lock( void ) {
        hipFree( mutex );
    }

    __device__ void lock( void ) {
        while( atomicCAS( mutex, 0, 1 ) != 0 );
    }

    __device__ void unlock( void ) {
        atomicExch( mutex, 0 );
    }
};

#endif
//end lock


void read_inputFile(string file_path, unsigned int num_hits);

// CUDA timer macros
hipEvent_t c_start, c_stop;

inline void start_time() {
    hipEventCreate(&c_start);
    hipEventCreate(&c_stop);
    hipEventRecord(c_start, 0);
}

inline float stop_time(const char *msg) {
  float elapsedTime = 0;
  hipEventRecord(c_stop, 0);
  hipEventSynchronize(c_stop);
  hipEventElapsedTime(&elapsedTime, c_start, c_stop);
  //printf("Time to %s: %.3f ms\n", msg, elapsedTime);
  hipEventDestroy(c_start);
  hipEventDestroy(c_stop);
  return elapsedTime;
}

__global__ void voteHoughSpace(float *dev_x_values, float *dev_y_values, float *dev_z_values, int *dev_accMat, float dtheta, float drho, float dphi){
  
  __shared__ float x_val;
  __shared__ float y_val;
  __shared__ float z_val;
   
  if(threadIdx.x == 0){
    x_val = dev_x_values[blockIdx.x];
    y_val = dev_y_values[blockIdx.x];
    z_val = dev_z_values[blockIdx.x];
  }
  
  __syncthreads();
  
  float R2 = x_val*x_val + y_val*y_val;
  float theta=acos(z_val/sqrt(R2+z_val*z_val));
  
  int ith=(int) (theta/dtheta)+0.5f;
  
  float sec=atan2(y_val,x_val);
  if (sec<0.f)
  {
    sec=2*M_PI+sec;
  }
  int isec=int(sec/2/M_PI*Nsec);
  
  int iphi = threadIdx.x;
  float phi=phimin+iphi*dphi;
  float rho=R2/2.f/(x_val*cos(phi)+y_val*sin(phi));
  int irho=(int)((rho-rhomin)/drho)+0.5f;
  
  int accu_index = (isec*(Ntheta*Nphi*Nrho))+((ith*Nphi*Nrho) +((iphi*Nrho)+irho));
  
  if (rho<=rhomax && rho>rhomin)
  {
    atomicAdd(&(dev_accMat[accu_index]),1);
  }
}

#ifndef PARALLEL_REDUX_MAX

#define get4DIndex(x,y,z,r) ((x)*(Ntheta*Nphi*Nrho))+(((y)*Nphi*Nrho) +(((z)*Nrho)+(r)))

__global__ void findRelativeMax(int *dev_accMat, struct track_param *dev_output, Lock mutex, unsigned int *NMrel){
  
  
  unsigned int isec = blockIdx.x;
  unsigned int ith = blockIdx.y / (Nrho/blockDim.y);
  unsigned int iphi = threadIdx.x;
  unsigned int irho = blockIdx.y % (Nrho/blockDim.y);
  
  unsigned int globalIndex = getGlobalIdx_2D_2D();
  unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
  
  /*__shared__ unsigned int local_NMrel;
  
  if(threadIdx.x == 0) local_NMrel = 0;
  __syncthreads();*/
  
  //check if it is a local maxima by verifying that it is greater then (>=) its neighboors
  
  //we must check from isec >= 0, ith >= 1, iphi >= 1, irho >= 1
  if((ith > 0) && (iphi > 0) && (irho > 0)){
    
    //each thread is assigned to one point of the accum. matrix:
    int acc= dev_accMat[get4DIndex(isec, ith, iphi, irho)];
    
    if (acc >= ac_soglia){
      
      if(acc > dev_accMat[get4DIndex(isec, ith,iphi, irho-1)] && acc >= dev_accMat[get4DIndex(isec, ith, iphi, irho+1)]){
	
	if(acc > dev_accMat[get4DIndex(isec, ith, iphi-1, irho)] && acc >= dev_accMat[get4DIndex(isec, ith, iphi+1, irho)]){
      
		/*atomicAdd(&local_NMrel, 1);
		
		if(threadIdx.x == 0){
		  mutex.lock();
		  *NMrel += local_NMrel;
		  mutex.unlock();
		}*/
		atomicAdd(NMrel, 1);
		
		//mutex.lock();
		dev_output[globalIndex].acc = acc;
		/*dev_output[globalIndex].isec = isec;
		dev_output[globalIndex].ith = ith;
		dev_output[globalIndex].iphi = iphi;
		dev_output[globalIndex].irho = irho;*/
		//mutex.unlock();
	}

      }
    }
    
    
  }               
}

#else

//NOTE: wrong approach to solve this problem
//TODO: improve as on slides
__global__ void reduceParallelMax(int *dev_accMat, int *dev_output, int *dev_maxRelOutput, unsigned int N){
  
  
  extern __shared__ int sdata[];
  
  int* max_sdata = (int *) sdata;
  int* relMax_sdata = (int *) &sdata[blockDim.x];
  
  //each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x; //local index
  //unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; //global index (1D grid - 1D block)
  unsigned int i = getGlobalIdx_2D_1D();
  
  if(i < N){ //check if thread is in data bounds
  
    max_sdata[tid] = dev_accMat[i];
    relMax_sdata[tid] = dev_accMat[i];
    __syncthreads();
    
    //do reduction in shared memory
    for(unsigned int s=1; s < blockDim.x; s*=2){
      if(tid % (2*s) == 0){ //it is for a different stride
	//atomicMax(&(max_sdata[tid]),max_sdata[tid+s]); //TODO: change without atomic
	max_sdata[tid] = (max_sdata[tid] > max_sdata[tid+s]) ? max_sdata[tid] : max_sdata[tid+s];
	__syncthreads();
      }
      __syncthreads();
      
    }
    
    //write results (now found in the first element of the array) for this block to global memory 
    //if(tid == 0) dev_output[blockIdx.x] = sdata[0];
    
    if(tid == 0) dev_output[blockIdx.x] = max_sdata[0]; //at sdata[0], we found the maximum
    
    if(relMax_sdata[tid] >= ac_soglia){ 
      dev_maxRelOutput[i] = relMax_sdata[tid];
    }else{
      dev_maxRelOutput[i] = 0;
    }

  }
}
#endif

void help(char* prog) {

  printf("Use %s [-l #loops] [-n #hitsToRead] [-h] \n\n", prog);
  printf("  -l loops        Number of executions (Default: 1).\n");
  printf("  -n hits         Number of hits to read from input file (Default: 236).\n");
  printf("  -h              This help.\n");

}

int main(int argc, char* argv[]){
  
  
    unsigned int N_LOOPS = 1;
    unsigned int N_HITS = 236;
    int c;
    
    //getting command line options
    while ( (c = getopt(argc, argv, "l:n:h")) != -1 ) {
      switch(c) {
	
	case 'n':
	  N_HITS = atoi(optarg);
	  break;
	  
	case 'l':
	  N_LOOPS = atoi(optarg);
	  break;
	case 'h':
	  help(argv[0]);
	  return 0;
	  break;
	default:
	  printf("Unkown option!\n");
	  help(argv[0]);
	  return 0;
      }
    }
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
#ifdef CUDA_MALLOCHOST_OUTPUT
      struct track_param *host_out_tracks;
      start_time();
      checkCudaErrors(hipHostMalloc((void **) &host_out_tracks, (sizeof(struct track_param)*(Nsec * Ntheta * Nphi * Nrho))));
      float init_outputMatrix = stop_time("init output matrix with hipHostMalloc");
      cout << "time to init output matrix (once): " << init_outputMatrix << endl;
#endif
  
    int *dev_accMat;
    float *dev_x_values;
    float *dev_y_values;
    float *dev_z_values;
    
    float *x_values_temp;
    float *y_values_temp;
    float *z_values_temp;
    
    //executions loop
    for(unsigned int loop = 0; loop < N_LOOPS; loop++){
      
      float timing[5];
      //float R = 0.f;
            
      // Inizializzo a zero le matrici
      memset(&acc_Mat, 0, (sizeof(int)*(Nsec*Ntheta*Nphi*Nrho)) );
      memset(&debug_accMat, 0, (sizeof(int)*(Nsec*Ntheta*Nphi*Nrho)) );
      //memset(&Max_rel, 0, (sizeof(int)*(Nsec*Ntheta*Nphi*Nrho)) );
      
      //alloc accumulator matrix on GPU
      start_time();
      checkCudaErrors(hipMalloc((void **) &dev_accMat, (sizeof(int)* (Nsec * Ntheta * Nphi * Nrho)) ));
      checkCudaErrors(hipMemset(dev_accMat, 0, (sizeof(int)*(Nsec*Ntheta*Nphi*Nrho))));
      timing[1] = stop_time("malloc dev_accMat and memset(0)");
      
      //riempi i valori dentro x_values , y_values , z_values
      read_inputFile("hits-5000.txt", N_HITS);
  //    read_inputFile("../datafiles/hits-1.txt");
      
      x_values_temp = (float*) malloc(sizeof(float)*x_values.size());
      y_values_temp =  (float*) malloc(sizeof(float)*y_values.size());
      z_values_temp = (float*)  malloc( sizeof(float)*z_values.size());
      
      for(unsigned int i = 0; i < x_values.size(); i++){
	x_values_temp[i] = x_values.at(i);
	y_values_temp[i] = y_values.at(i);
	z_values_temp[i] = z_values.at(i);
      }
      
      start_time();
      checkCudaErrors(hipMalloc((void **) &dev_x_values, sizeof(float)*x_values.size()));
      checkCudaErrors(hipMalloc((void **) &dev_y_values, sizeof(float)*y_values.size()));
      checkCudaErrors(hipMalloc((void **) &dev_z_values, sizeof(float)*z_values.size()));
      checkCudaErrors(hipMemcpy(dev_x_values, x_values_temp, sizeof(float)*x_values.size(), hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(dev_y_values, y_values_temp, sizeof(float)*y_values.size(), hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(dev_z_values, z_values_temp, sizeof(float)*z_values.size(), hipMemcpyHostToDevice));
      timing[0] = stop_time("Input malloc and copy HtoD");
      
      start_time();
      voteHoughSpace <<<x_values.size(), Nphi>>> (dev_x_values, dev_y_values, dev_z_values, dev_accMat, dtheta, drho, dphi); //assumes that Nphi == Nrho
      timing[2] = stop_time("Vote");
#ifdef VERBOSE_DUMP     
      checkCudaErrors(hipMemcpy((void *) &debug_accMat, dev_accMat, (sizeof(int)*(Nsec*Ntheta*Nphi*Nrho)), hipMemcpyDeviceToHost));
#endif
      for(unsigned int i = 0; i < x_values.size(); i++){
	  //cout << x_values.at(i) << " - ";
	  //cout << y_values.at(i) << endl;
	  
	  float R2=x_values.at(i)*x_values.at(i)+y_values.at(i)*y_values.at(i);
	  float theta=acos(z_values.at(i)/sqrt(R2+z_values.at(i)*z_values.at(i)));
	  int ith=(int) (theta/dtheta)+0.5f;
	  
	  float sec=atan2(y_values.at(i),x_values.at(i));
	  if (sec<0.f)
	  {
	      sec=2*M_PI+sec;
	  }
	  int isec=int(sec/2/M_PI*Nsec);
	  
	  for(int iphi = 0; iphi < Nphi; iphi++){
	      float phi=phimin+iphi*dphi;
	      float rho=R2/2.f/(x_values.at(i)*cos(phi)+y_values.at(i)*sin(phi));
	      int irho=(int)((rho-rhomin)/drho)+0.5f;
	      if (rho<=rhomax && rho>rhomin)
	      {
		  acc_Mat[isec][ith][iphi][irho]++;
	      }
	  }
      }
      
#ifdef VERBOSE_DUMP
      //check
      unsigned int corretto = 0;
      unsigned int errore = 0;
      unsigned int letto = 0;
      for(unsigned int isec = 0; isec < Nsec; isec++){
	  
	  for(unsigned int ith = 0; ith < Ntheta; ith++){
	      
	      for(unsigned int iphi = 0; iphi < Nphi; iphi++){
		  
		  for(unsigned int irho = 0; irho < Nrho; irho++){
		    
		    if(acc_Mat[isec][ith][iphi][irho] != debug_accMat[isec][ith][iphi][irho]){
		    printf("diverso acc_Mat[%d][%d][%d][%d] %d - debug_accMat[%d][%d][%d][%d] %d \n", isec, ith, iphi, irho, acc_Mat[isec][ith][iphi][irho],
		      isec, ith, iphi, irho, debug_accMat[isec][ith][iphi][irho]);
		      errore++;
		    }else corretto++;
		    letto++;
		  }
	      }
	  }
      }
      printf("corretti %d sbaglati %d; letti %d\n", corretto, errore, letto);
#endif
      
      checkCudaErrors(hipFree(dev_x_values));
      checkCudaErrors(hipFree(dev_y_values));
      checkCudaErrors(hipFree(dev_z_values));
      
      free(x_values_temp);
      free(y_values_temp);
      free(z_values_temp);
      
      x_values.clear();
      y_values.clear();
      z_values.clear();
      
      //trova il massimo relativo
      unsigned int host_NMrel = 0;
      
      // --- Prendiamo le informazioni specifiche della GPU per la divisione del lavoro appropriata
      unsigned int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

#ifndef PARALLEL_REDUX_MAX
      
      struct track_param *dev_indexOutput;
      Lock my_lock;
      
      unsigned int *NMrel;
      
      start_time();   
      checkCudaErrors(hipMalloc((void **) &NMrel, (sizeof(unsigned int))));
      checkCudaErrors(hipMemset(NMrel, 0, sizeof(unsigned int)));
      
#ifdef CUDA_MANAGED_TRANSFER
      int cudaVer = 0;
      hipRuntimeGetVersion(&cudaVer);
      if(cudaVer >= 6000){
	checkCudaErrors(hipMallocManaged(&dev_indexOutput,(sizeof(struct track_param)* (Nsec * Ntheta * Nphi * Nrho)) ));
      }else{
#endif
	
      checkCudaErrors(hipMalloc((void **) &dev_indexOutput, (sizeof(struct track_param)* (Nsec * Ntheta * Nphi * Nrho)) ));
      
#ifdef CUDA_MANAGED_TRANSFER
      }
#endif
      
      checkCudaErrors(hipMemset(dev_indexOutput, -1, (sizeof(struct track_param)* (Nsec * Ntheta * Nphi * Nrho))));
      
      timing[1] += stop_time("malloc dev_indexOutput+NMrel and memset");
      
      // dividiamo adeguatamente il lavoro
      // in base al numero massimo di thread disponibili in un singolo thread-block
      unsigned int dim_x_block = Nphi;
      unsigned int dim_y_block = maxThreadsPerBlock/dim_x_block;
      unsigned int dim_x_grid = Nsec;
      unsigned int dim_y_grid = Ntheta * (Nrho/dim_y_block);
      
      dim3 grid(dim_x_grid, dim_y_grid);
      dim3 block(dim_x_block, dim_y_block);
      
      start_time();
      findRelativeMax <<<grid, block>>> (dev_accMat, dev_indexOutput, my_lock, NMrel);
      timing[3] = stop_time("Max. Relative");
      
      start_time();
#ifdef CUDA_MALLOCHOST_OUTPUT
      checkCudaErrors(hipMemcpy((void *) host_out_tracks, dev_indexOutput, (sizeof(int)* (Nsec * Ntheta * Nphi * Nrho)), hipMemcpyDeviceToHost));
#else
      checkCudaErrors(hipMemcpy((void *) &host_out_tracks, dev_indexOutput, (sizeof(int)* (Nsec * Ntheta * Nphi * Nrho)), hipMemcpyDeviceToHost));
#endif
      checkCudaErrors(hipMemcpy((void *) &host_NMrel, NMrel, (sizeof(int)), hipMemcpyDeviceToHost));
      timing[4] = stop_time("Copy results DtoH");

#ifdef VERBOSE_DUMP
      cout << "NMrel from GPU "<< host_NMrel << endl;

      unsigned int ntracks = 0;
      
      for(unsigned int i = 0; ((i < (Nsec * Ntheta * Nphi * Nrho)) && (ntracks < host_NMrel)); i++){
	
	if(host_out_tracks[i].acc > -1){
	  cout << "track " << ntracks << " acc value = " << host_out_tracks[i].acc << " [" << i << "]" << endl;
	  ntracks++;    
	}
      }
#endif
      //free mem
      checkCudaErrors(hipFree(dev_indexOutput));
      checkCudaErrors(hipFree(NMrel));

      //print timing results with this format:
      // NHIT HtoD_input MEMSET_cumulative VOTE MAX_REL DtoH_output
      cout << N_HITS << " " << timing[0] << " " << timing[1] << " " << timing[2] << " " << timing[3] << " " << timing[4] << endl; 
      
      
#else
      
#define SET_GRID_DIM(npoints, threadsPerBlock) ceil((npoints+((threadsPerBlock)-1))/(threadsPerBlock))
      
      unsigned int half_grid = SET_GRID_DIM((Nsec*Ntheta*Nphi*Nrho), maxThreadsPerBlock)/2;
      
      dim3 grid(half_grid, 2);
      
      unsigned int n_blocks = half_grid * 2;
      
      int * dev_maxBlockOutput;
      checkCudaErrors(hipMalloc((void **) &dev_maxBlockOutput, (sizeof(int) * n_blocks)));
      int * dev_maxRelOutput;
      checkCudaErrors(hipMalloc((void **) &dev_maxRelOutput, (sizeof(int) * (Nsec*Ntheta*Nphi*Nrho))));
      
      reduceParallelMax<<<grid, maxThreadsPerBlock, 2*(maxThreadsPerBlock*sizeof(int))>>>(dev_accMat, dev_maxBlockOutput, dev_maxRelOutput, (Nsec*Ntheta*Nphi*Nrho));
      
      int *host_maxBlockOutput = (int *) malloc((sizeof(int)* n_blocks));
      checkCudaErrors(hipMemcpy(host_maxBlockOutput, dev_maxBlockOutput, (sizeof(int) * n_blocks), hipMemcpyDeviceToHost));
      
      int *host_maxRelOutput = (int *) malloc((sizeof(int)* (Nsec*Ntheta*Nphi*Nrho)));
      checkCudaErrors(hipMemcpy(host_maxRelOutput, dev_maxRelOutput, (sizeof(int) * (Nsec*Ntheta*Nphi*Nrho)), hipMemcpyDeviceToHost));
      
      unsigned int debug = 0;
      
      for(unsigned int i = 0; i < n_blocks; i++){
	
	if(host_maxBlockOutput[i] != 0){
	  cout << "block " << i << " max: " << host_maxBlockOutput[i] << " [" << i*maxThreadsPerBlock << "]" << endl;
	  host_NMrel++;
	}
	
	unsigned int found = 0;
	
	for(unsigned int y = 0; y < maxThreadsPerBlock; y++){
	  unsigned int globalIndex = (y+(i*maxThreadsPerBlock));
	  if((host_maxRelOutput[globalIndex] != 0)) {
	    cout << "out["<< globalIndex << "]="<< host_maxRelOutput[globalIndex]<< " ";
	    found++; debug++;
	  }
	}
	if(found > 0) cout << " (block "<< i << ")" << endl << endl;
	
      }
      
      
      /*for(unsigned int i = 0; i < (Nsec*Ntheta*Nphi*Nrho); i += maxThreadsPerBlock){
	
	if(host_maxBlockOutput[i] != 0) cout << "block" << i/maxThreadsPerBlock << " max: " << host_maxBlockOutput[i] << " [" << i << "]" << endl;
	
	unsigned int found = 0;
	
	for(unsigned int y = 0; y < (maxThreadsPerBlock); y++){ // check relative maxima
	  if((host_maxRelOutput[i+y] != 0)){ cout << "out["<< i+y << "]="<< host_maxRelOutput[i+y]<< " "; found++; host_NMrel++;}
	}
	if(found > 0) cout << endl << endl;
      }*/
      
      cout << "NMrel from GPU "<< host_NMrel << " " << debug << endl;
      
      hipFree(dev_maxBlockOutput);
      hipFree(dev_maxRelOutput);
      
      free(host_maxBlockOutput);
      free(host_maxRelOutput);
      
#endif  
      
      host_NMrel = 0;
      
      int accumax = -1;
      int iphiMax = 0;
      int irhoMax = 0;
      int ithMax = 0;
      int isecMax = 0;
      
      
      for(unsigned int isec = 0; isec < Nsec; isec++){
	  
	  for(unsigned int ith = 1; ith < Ntheta; ith++){
	      
	      for(unsigned int iphi = 1; iphi < Nphi; iphi++){
		  
		  for(unsigned int irho = 1; irho < Nrho; irho++){
		      
		      float acc=acc_Mat[isec][ith][iphi][irho];
		      if (acc >= ac_soglia){
			  if (acc > accumax){
			      accumax=acc;
			  }
			  /*if (acc>acc_Mat[isec][ith-1][iphi][irho] && acc >= acc_Mat[isec][ith+1][iphi][irho]){
			      if (acc>acc_Mat[isec][ith][iphi-1][irho-1] && acc >= acc_Mat[isec][ith][iphi-1][irho+1]){ //TODO: chiedi a Lorenzo perché [iphi+1][irho+1] invece di [iphi-1][irho+1]
				  if (acc>acc_Mat[isec][ith][iphi][irho-1] && acc >= acc_Mat[isec][ith][iphi][irho+1]){
				      if (acc>acc_Mat[isec][ith][iphi+1][irho-1] && acc >= acc_Mat[isec][ith][iphi+1][irho+1]){*/
			  
			  if(acc > acc_Mat[isec][ith][iphi][irho-1] && acc >= acc_Mat[isec][ith][iphi][irho+1]){
			    if(acc > acc_Mat[isec][ith][iphi-1][irho] && acc >= acc_Mat[isec][ith][iphi+1][irho]){
					  //if (acc>=acc_Mat[isec][ith][iphi+1][irho] ){
					      accumax = acc_Mat[isec][ith][iphi+1][irho];
					      //Max_rel[isec][ith][iphi+1][irho]=1;
					      host_NMrel++;
					      ithMax=ith;
					      irhoMax=irho;
					      iphiMax=iphi;
					      isecMax=isec+1;
					      float t_th=(thetamin+ithMax*dtheta)*360.f/M_PI;
					      float t_rho=rhomin+irhoMax*drho;
					      float t_phi=phimin+iphiMax*dphi;
					      //float q=t_rho/sin(t_phi);
					      //float xm=-1/tan(t_phi);
					      //cout << acc <<" "<< t_rho <<" "<< t_phi << " " << isecMax << endl;
					      
					  //}
				      //}
				  //}
			      }
			  }
		      }
		  }
	      }
	  }
      }
#ifdef VERBOSE_DUMP
      cout << "NMrel from CPU "<< host_NMrel << endl;
#endif
      checkCudaErrors(hipFree(dev_accMat));
      
      
  }
  
#ifdef CUDA_MALLOCHOST_OUTPUT      
  checkCudaErrors(hipHostFree(host_out_tracks));
#endif
    
    return 0;
}

/*****************************
 * file opener
 *****************************/


void read_inputFile(string file_path, unsigned int num_hits)
{
    
    ifstream input_f;
    
    string line;
    string value;
    
    stringstream ss;
    unsigned int val_iter;
    
    unsigned int line_read = 0;
    
    input_f.open(file_path.c_str());
    
    if (input_f.is_open())
    {
        while ( getline (input_f,line) && (line_read < num_hits) )
        {
            val_iter = 0;
            ss.str(line);
            //prendiamo dati direttamente dal file ASCII in input
            while(ss >> value){
                //i valori che ci interessano sono X, Y e Z
                if (val_iter == 0) x_values.push_back(atof(value.c_str()));
                else if (val_iter == 1) y_values.push_back(atof(value.c_str()));
                else if (val_iter == 2) z_values.push_back(atof(value.c_str()));
                val_iter++;
                
            }
            ss.clear();
	    line_read++;
        }
        input_f.close();
    }
    
    
    
    
}

