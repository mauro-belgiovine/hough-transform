#include "hip/hip_runtime.h"
// **********************************************************************************
//
// BSD License.
// This file is part of a Hough Transformation tutorial,
// see: http://www.keymolen.com/2013/05/hough-transformation-c-implementation.html
//
// Copyright (c) 2013, Bruno Keymolen, email: bruno.keymolen@gmail.com
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
// Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.
// Redistributions in binary form must reproduce the above copyright notice, this
// list of conditions and the following disclaimer in the documentation and/or other
// materials provided with the distribution.
// Neither the name of "Bruno Keymolen" nor the names of its contributors may be
// used to endorse or promote products derived from this software without specific
// prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
// IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
// INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
// NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
// WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// **********************************************************************************

#include "hough.h"
#include "simpleIndexing.cu"
#include <cmath>
#include <iostream>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper utility functions 

#define DEG2RAD 0.017453293f

#define SHARED_MEM_BANKS 32
#define SET_GRID_DIM(npoints, threadsPerBlock) ceil((npoints+(threadsPerBlock-1))/threadsPerBlock)
#define BLOCK_DIM 8
#define BLOCK_DIM_Y (SHARED_MEM_BANKS/BLOCK_DIM)

using namespace std;

/* CUDA functions definitions */

// CUDA timer macros
hipEvent_t c_start, c_stop;

inline void start_time() {
    hipEventCreate(&c_start);
    hipEventCreate(&c_stop);
    hipEventRecord(c_start, 0);
}

inline float stop_time(const char *msg) {
  float elapsedTime = 0;
  hipEventRecord(c_stop, 0);
  hipEventSynchronize(c_stop);
  hipEventElapsedTime(&elapsedTime, c_start, c_stop);
  //if ( VERBOSE )
  printf("Time to %s: %.3f ms\n", msg, elapsedTime);
  hipEventDestroy(c_start);
  hipEventDestroy(c_stop);
  return elapsedTime;
}


/* getPixels(): kernel used to making an array of all pixels that need to be processed

NOTE: dovendo accedere allo stesso indirizzo di memoria __shared__, dobbiamo garantire un accesso
privo di bank-conflict, per evitare perdita di informazioni dovute all'accesso concorrente: 
per la memoria __shared__, ciò è garantito solo in un warp (32 thread) per device 2.x (16 thread per 1.x) che accede allo stesso bank (32 bit) di memoria
grazie alle primitive di "broadcast"

The fast case:
- If all threads of a half-warp access different banks, there is no bank conflict
- ** If all threads of a half-warp read the identical address, there is no bank conflict (broadcast) **
The slow case:
- Bank Conflict: multiple threads in the same half-warp access the same bank
- Must serialize the accesses
- Cost = max # of simultaneous accesses to a single bank

FROM -> http://on-demand.gputechconf.com/gtc-express/2011/presentations/NVIDIA_GPU_Computing_Webinars_CUDA_Memory_Optimization.pdf

PSEUDO CODE:
1 pixel_value = image[x,y]
2 if(pixel_value > threshold) {
3 	do {
4 		index++
5 		SMEM_index = index
6 		SMEM_array[index] = (x,y)
7 	} while(SMEM_array[index] != (x,y))
8 }
9 index = SMEM_index
*/

__global__ void getPixels(unsigned char* dev_img, unsigned int *dev_globalPixelArray, unsigned int *dev_globalPixelCount,  int w, int h){
  
  //calculate index which this thread have to process
  unsigned int index = getGlobalIdx_2D_2D();
  unsigned int pixel_count = 0;
  __shared__ unsigned int sh_pixel_count;
  __shared__ unsigned int sh_pixel_array[BLOCK_DIM*BLOCK_DIM];
  
  //__shared__ unsigned int sh_entered[BLOCK_DIM*BLOCK_DIM];
  
  
  //sh_entered[(threadIdx.y * blockDim.x) + threadIdx.x] = 0;
  
  unsigned int blockIndex = (threadIdx.y * blockDim.x) + threadIdx.x;
  
  if(blockIndex == 0) sh_pixel_count = 0;
  
  
  //check index is in image bounds
  if(index < (w*h)){
    
    if( dev_img[index] > 250 ){ //se il punto è bianco (val in scala di grigio > 250)
      
      //sh_entered[(threadIdx.y * blockDim.x) + threadIdx.x] += 1;
      
      do{
	pixel_count++;
	sh_pixel_count = pixel_count;
	sh_pixel_array[pixel_count] = index;
	__syncthreads();
      }while(sh_pixel_array[pixel_count] != index );
    }
    
    pixel_count = sh_pixel_count;
  }
  
 
  
  unsigned int blockId = (blockIdx.y * gridDim.x) + blockIdx.x;
  
  //First one thread in each thread block
  if((threadIdx.x == 0) && (threadIdx.y == 0)){
    //add the sum of all pixels collected in this thread-block
    dev_globalPixelCount[blockId] = pixel_count;
    
    /*for(unsigned int x = 0; x < BLOCK_DIM; x++){ //loop on threadIdx.x
      for(unsigned int y = 0; y < BLOCK_DIM; y++){ //loop on threadIdx.y
	dev_globalPixelCount[blockId] += sh_entered[(y * blockDim.x) + x];
      }
    }*/
    
    
    
    //copy in the global array each pixel to be processed
    /*for(unsigned int i = 0; i < pixel_count; i++){
      dev_globalPixelArray[i] = sh_pixel_array[i]; // <--------- TODO: trovare modo efficiente per salvare la lista dei punti da processare
    }*/
  }
}




//every CUDA Thread works processes a point of the input image
__global__ void CudaTransform(unsigned char* dev_img, unsigned int *dev_accu, int w, int h){
  
  
  //calculate index which this thread have to process
  unsigned int index = getGlobalIdx_2D_2D();
  
  //check index is in image bounds
  if(index < (w*h)){
    //calculate params
    float hough_h = ((sqrt(2.0) * (float)(h>w?h:w)) / 2.0);
	    
    float center_x = w/2;
    float center_y = h/2;
      
    //calculate coordinates for corresponding index in entire image
    int x = index % w;
    int y = index / w;
    
    if( dev_img[index] > 250 ){ //se il punto è bianco (val in scala di grigio > 250)
      for(int t=0;t<180;t++){ //plot dello spazio dei parametri da 0° a 180° (sist. polare)
	
	float r = ( ((float)x - center_x) * cos((float)t * DEG2RAD)) + (((float)y - center_y) * sin((float)t * DEG2RAD));
	
	//dev_accu[ (int)((round(r + hough_h) * 180.0)) + t]++;
	atomicAdd(&(dev_accu[ (int)((round(r + hough_h) * 180.0)) + t]), 1);
	
      }
    }
  }
  
}

namespace keymolen {

	Hough::Hough():_accu(0), _accu_w(0), _accu_h(0), _img_w(0), _img_h(0)
	{

	}

	Hough::~Hough() {
		if(_accu)
			free(_accu);
	}


	int Hough::Transform(unsigned char* img_data, int w, int h)
	{
	  
		
		_img_w = w;
		_img_h = h;

		//Create the accu
		double hough_h = ((sqrt(2.0) * (double)(h>w?h:w)) / 2.0);
		_accu_h = hough_h * 2.0; // -r -> +r
		_accu_w = 180;

		_accu = (unsigned int*)calloc(_accu_h * _accu_w, sizeof(unsigned int));

		double center_x = w/2;
		double center_y = h/2;

		start_time();
		
		unsigned int total_processed_pixels = 0;

		for(int y=0;y<h;y++)
		{
			for(int x=0;x<w;x++)
			{
				if( img_data[ (y*w) + x] > 250 )
				{
				  total_processed_pixels++;
					for(int t=0;t<180;t++)
					{
						double r = ( ((double)x - center_x) * cos((double)t * DEG2RAD)) + (((double)y - center_y) * sin((double)t * DEG2RAD));
						_accu[ (int)((round(r + hough_h) * 180.0)) + t]++; 
					}
				}
			}
		}
		
		cout << "Total processed pixels " << total_processed_pixels << endl;
		
		stop_time("CPU Transform");
		return 0;
	}
	
	int Hough::Transform_GPU(unsigned char* img_data, int w, int h){
	  
	  
	  _img_w = w;
	  _img_h = h;

	  //Create the accu
	  double hough_h = ((sqrt(2.0) * (double)(h>w?h:w)) / 2.0);
	  _accu_h = hough_h * 2.0; // -r -> +r
	  _accu_w = 180;
	  _accu = (unsigned int*)calloc(_accu_h * _accu_w, sizeof(unsigned int));
	  
	  unsigned char *dev_img;
	  unsigned int *dev_accu;
	  
	  
	  
	  checkCudaErrors(hipMalloc((void **) &dev_img, (sizeof(char)*w*h)));
	  checkCudaErrors(hipMalloc((void **) &dev_accu, (sizeof(unsigned int) * _accu_w * _accu_h)));
	  checkCudaErrors(hipMemset(dev_accu, 0, (sizeof(unsigned int) * _accu_w * _accu_h)));
	  
	  //copy data on device
	  checkCudaErrors(hipMemcpy(dev_img, img_data, (sizeof(char)*w*h), hipMemcpyHostToDevice));
	  
	  
	  //launch kernel
	  dim3 block(BLOCK_DIM, 4);
	  dim3 grid(SET_GRID_DIM(w,BLOCK_DIM), SET_GRID_DIM(h,4));
	  start_time();
	  CudaTransform <<< grid, block >>> (dev_img, dev_accu, w, h);
	  stop_time("GPU Transform");
	  
	  //copy back results
	  checkCudaErrors(hipMemcpy(_accu, dev_accu, (sizeof(unsigned int) * _accu_w * _accu_h), hipMemcpyDeviceToHost));
	  
	  hipFree(dev_img);
	  hipFree(dev_accu);
	  return 0;
	}
	
	int Hough::Transform_GPUFast(unsigned char* img_data, int w, int h){
	  
	  _img_w = w;
	  _img_h = h;

	  //Create the accu
	  double hough_h = ((sqrt(2.0) * (double)(h>w?h:w)) / 2.0);
	  _accu_h = hough_h * 2.0; // -r -> +r
	  _accu_w = 180;
	  _accu = (unsigned int*)calloc(_accu_h * _accu_w, sizeof(unsigned int));
	  
	  unsigned char *dev_img;
	  unsigned int *dev_globalPixelArray;	//it will contain only pixels that have to be processed
	  unsigned int *dev_globalPixelCount;	//it will hold number of pixels that have to be processed per each thread-BLOCK
	  
	  checkCudaErrors(hipMalloc((void **) &dev_img, (sizeof(char) * w * h)));
	  //copy data on device
	  checkCudaErrors(hipMemcpy(dev_img, img_data, (sizeof(char)*w*h), hipMemcpyHostToDevice));
	  
	  checkCudaErrors(hipMalloc((void **) &dev_globalPixelArray, (sizeof(unsigned int) * w * h)));
	  checkCudaErrors(hipMalloc((void **) &dev_globalPixelCount, (sizeof(unsigned int) * SET_GRID_DIM(w,BLOCK_DIM) * SET_GRID_DIM(h,BLOCK_DIM_Y))));
	  checkCudaErrors(hipMemset(dev_globalPixelCount, 0 , (sizeof(unsigned int) * SET_GRID_DIM(w,BLOCK_DIM) * SET_GRID_DIM(h,BLOCK_DIM_Y))));
	  
	  dim3 block(BLOCK_DIM, BLOCK_DIM_Y);
	  dim3 grid(SET_GRID_DIM(w,BLOCK_DIM), SET_GRID_DIM(h,BLOCK_DIM_Y));
	  
	  start_time();
	  getPixels <<<grid, block>>> (dev_img, dev_globalPixelArray, dev_globalPixelCount, w, h);
	  stop_time("Fast GPU Transform");
	  
	  unsigned int *PixelCount = (unsigned int *) malloc(sizeof(unsigned int) * SET_GRID_DIM(w,BLOCK_DIM) * SET_GRID_DIM(h,BLOCK_DIM_Y));
	  checkCudaErrors(hipMemcpy(PixelCount, dev_globalPixelCount, (sizeof(unsigned int) * SET_GRID_DIM(w,BLOCK_DIM) * SET_GRID_DIM(h,BLOCK_DIM_Y)), hipMemcpyDeviceToHost));

	  unsigned int total_pix = 0;
	  for(unsigned int i = 0; i < (SET_GRID_DIM(w,BLOCK_DIM) * SET_GRID_DIM(h,BLOCK_DIM_Y)); i++){
	    cout << "block ID " << i << "=" << PixelCount[i] << " ";
	    total_pix += PixelCount[i];
	    if ((i % 5 ) == 0){
	      cout << endl;
	    }
	  }
	  cout << "total pixels " << total_pix << endl;
	  
	  return 0;
	}

	std::vector< std::pair< std::pair<int, int>, std::pair<int, int> > > Hough::GetLines(int threshold)
	{
		std::vector< std::pair< std::pair<int, int>, std::pair<int, int> > > lines;

		if(_accu == 0)
			return lines;

		for(int r=0;r<_accu_h;r++)
		{
			for(int t=0;t<_accu_w;t++)
			{
				if((int)_accu[(r*_accu_w) + t] >= threshold)
				{
					//Is this point a local maxima (9x9)
					int max = _accu[(r*_accu_w) + t];
					for(int ly=-4;ly<=4;ly++)
					{
						for(int lx=-4;lx<=4;lx++)
						{
							if( (ly+r>=0 && ly+r<_accu_h) && (lx+t>=0 && lx+t<_accu_w)  )
							{
								if( (int)_accu[( (r+ly)*_accu_w) + (t+lx)] > max )
								{
									max = _accu[( (r+ly)*_accu_w) + (t+lx)];
									ly = lx = 5;
								}
							}
						}
					}
					if(max > (int)_accu[(r*_accu_w) + t])
						continue;


					int x1, y1, x2, y2;
					x1 = y1 = x2 = y2 = 0;

					if(t >= 45 && t <= 135)
					{
						//y = (r - x cos(t)) / sin(t)
						x1 = 0;
						y1 = ((double)(r-(_accu_h/2)) - ((x1 - (_img_w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (_img_h / 2);
						x2 = _img_w - 0;
						y2 = ((double)(r-(_accu_h/2)) - ((x2 - (_img_w/2) ) * cos(t * DEG2RAD))) / sin(t * DEG2RAD) + (_img_h / 2);
					}
					else
					{
						//x = (r - y sin(t)) / cos(t);
						y1 = 0;
						x1 = ((double)(r-(_accu_h/2)) - ((y1 - (_img_h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (_img_w / 2);
						y2 = _img_h - 0;
						x2 = ((double)(r-(_accu_h/2)) - ((y2 - (_img_h/2) ) * sin(t * DEG2RAD))) / cos(t * DEG2RAD) + (_img_w / 2);
					}

					lines.push_back(std::pair< std::pair<int, int>, std::pair<int, int> >(std::pair<int, int>(x1,y1), std::pair<int, int>(x2,y2)));

				}
			}
		}

		std::cout << "lines: " << lines.size() << " " << threshold << "; img dim: w=" << _img_w << " h=" << _img_h << std::endl;
		return lines;
	}

	const unsigned int* Hough::GetAccu(int *w, int *h)
	{
		*w = _accu_w;
		*h = _accu_h;

		return _accu;
	}
}
